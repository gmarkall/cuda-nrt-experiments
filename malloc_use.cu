
#include <hip/hip_runtime.h>
__global__ void use_malloc(int **p)
{
  *p = static_cast<int*>(malloc(sizeof(int)));
}

__global__ void use_free(int *p)
{
  free(p);
}
