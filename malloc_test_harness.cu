#include <iostream>
#include <hip/hip_runtime.h>

__global__ void allocate(int **p)
{
  *p = static_cast<int*>(malloc(sizeof(int)));
}

__global__ void deallocate(int *p)
{
  free(p);
}

__managed__ int *p;

int main(int argc, char **argv)
{
  hipError_t err;
  hipSetDevice(0);
  constexpr size_t ONE_MEGABYTE = 1024 * 1024;
  hipDeviceSetLimit(hipLimitMallocHeapSize, ONE_MEGABYTE);
  std::cout << "Allocating... ";
  allocate<<<1, 1>>>(&p);
  err = hipDeviceSynchronize();
  std::cout << "result: '" << hipGetErrorString(err) << "'" << std::endl;
  std::cout << "Deallocating... ";
  deallocate<<<1, 1>>>(p);
  err = hipDeviceSynchronize();
  std::cout << "result: '" << hipGetErrorString(err) << "'" << std::endl;
  return 0;
}
