#include "hip/hip_runtime.h"
#include "nrt.cuh"

extern "C"
__device__ int init_memsys(void* dummy_return, uint64_t memsys_ptr, bool stats_enabled)
{
  TheMSys = reinterpret_cast<NRT_MemSys*>(memsys_ptr);

  TheMSys->allocator.malloc = static_cast<NRT_malloc_func>(malloc_wrapper);
  TheMSys->allocator.realloc = static_cast<NRT_realloc_func>(nullptr);
  TheMSys->allocator.free = static_cast<NRT_free_func>(free_wrapper);

  TheMSys->stats.enabled = stats_enabled;
  TheMSys->stats.alloc = 0;
  TheMSys->stats.free = 0;
  TheMSys->stats.mi_alloc = 0;
  TheMSys->stats.mi_free = 0;

  return 0;
}

extern "C"
__device__ int sizeof_memsys(size_t *size)
{
  *size = sizeof(NRT_MemSys);
  return 0;
}

__global__ void init_stats(NRT_Stats *stats, bool stats_enabled)
{
  memsys_stats->enabled = stats_enabled;
  memsys_stats->alloc = 0;
  memsys_stats->free = 0;
  memsys_stats->mi_alloc = 0;
  memsys_stats->mi_free = 0;
  *stats = &memsys_stats;
}
